/**
 * @ Author: Kai Xu
 * @ Create Time: 2020-06-03 17:57:42
 * @ Modified by: Kai Xu
 * @ Modified time: 2020-06-05 22:39:00
 * @ Description:
 */
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime.h>
 
 #include "quadtree.hpp"

 namespace ms
{
    void quadtree_cpy_trees_gpu_cpu_cuda(const qt_tree_t* src, qt_tree_t* dst, int num_blocks) {
        if(DEBUG) { printf("[DEBUG] quadtree_cpy_trees_gpu_cpu\n"); }
        device_to_host(src, dst, num_blocks * N_TREE_INTS);
    }
}